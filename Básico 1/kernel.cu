﻿// Includes necesarios
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 8 // Definimos el tamaño del array (en este caso, 8 elementos)

// Función principal que se ejecuta en el host (CPU)
int main(int argc, char** argv)
{
    // Declaración de punteros para los arrays en memoria del host y device
    float* hst_A, * hst_B; // hst_A y hst_B en el host (CPU)
    float* dev_A, * dev_B; // dev_A y dev_B en el device (GPU)

    // Reserva de memoria en el host para los arrays
    hst_A = (float*)malloc(N * sizeof(float)); // Reserva para hst_A
    hst_B = (float*)malloc(N * sizeof(float)); // Reserva para hst_B

    // Reserva de memoria en el device para los arrays
    hipMalloc((void**)&dev_A, N * sizeof(float)); // Reserva para dev_A en la GPU
    hipMalloc((void**)&dev_B, N * sizeof(float)); // Reserva para dev_B en la GPU

    // Inicialización de los datos en el array hst_A con números aleatorios entre 0 y 1
    srand((int)23412345); // Semilla para generar números aleatorios
    for (int i = 0; i < N; i++)
    {
        hst_A[i] = (float)rand() / RAND_MAX;  // Genera números aleatorios entre 0 y 1
    }

    // Mostrar los datos generados en hst_A (entrada)
    printf("ENTRADA (hst_A):\n");
    for (int i = 0; i < N; i++)
    {
        printf("%.2f ", hst_A[i]); // Imprime con 2 decimales
    }
    printf("\n");

    // Transferencia de datos desde el host (hst_A) al device (dev_A)
    hipMemcpy(dev_A, hst_A, N * sizeof(float), hipMemcpyHostToDevice);

    // Copia de datos dentro del device: de dev_A a dev_B
    hipMemcpy(dev_B, dev_A, N * sizeof(float), hipMemcpyDeviceToDevice);

    // Transferencia de datos desde el device (dev_B) al host (hst_B)
    hipMemcpy(hst_B, dev_B, N * sizeof(float), hipMemcpyDeviceToHost);

    // Mostrar los datos copiados en hst_B (salida)
    printf("SALIDA (hst_B):\n");
    for (int i = 0; i < N; i++)
    {
        printf("%.2f ", hst_B[i]); // Imprime con 2 decimales
    }
    printf("\n");

    // Liberación de memoria en el host y en el device para evitar problemas de fuga de memoria.
    free(hst_A); // Liberamos la memoria reservada para hst_A en el host
    free(hst_B); // Liberamos la memoria reservada para hst_B en el host
    hipFree(dev_A); // Liberamos la memoria reservada para dev_A en el device
    hipFree(dev_B); // Liberamos la memoria reservada para dev_B en el device

    // Mostrar la fecha y hora de ejecución del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha)); // Muestra la fecha y hora actual
    printf("<pulsa [INTRO] para finalizar>");
    getchar(); // Pausa antes de finalizar

    return 0;
}
